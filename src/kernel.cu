#include "hip/hip_runtime.h"
#include "../include/kernel.cuh"

__host__ __device__ int8_t quantize(int8_t a, int8_t b) {
    /* quantizes a pair of values to fit two 4-bit signed integers into a single byte */
    
    if (a > MAX_4BIT || a < MIN_4BIT || b > MAX_4BIT || b < MIN_4BIT) {
        printf("ERROR: Input integers must fit in 4 bits\n");
        return 0x80; // Set highest bit to indicate error
    }

    // Mask for value of signed ints: 2^3 & 2^8 ( [0]000[0]000 )
    int8_t a_val = a & 0x08;
    int8_t b_val = (b >> 4) & 0x08;

    // Masking for absolute value
    int8_t a_mask = a >> 7;
    a = (a ^ a_mask) - a_mask;
    int8_t b_mask = b >> 7;
    b = (b ^ b_mask) - b_mask;

    // transalte 4bit ints to left & right half of 8-bit int
    a_val += (a & 7) * 16;
    b_val += b & 7;

    return int8_t(a_val + b_val);
}

__global__ void quantize_array_kernel(const int8_t* input, int8_t* output, int n) {
    // Each thread will process two elements, so we calculate the index accordingly
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (2 * idx + 1 < n) { // Make sure we have both elements
        int8_t a = input[2 * idx];
        int8_t b = input[2 * idx + 1];
        output[idx] = quantize(a, b);
    }
}

void quantize_array(const int8_t* h_input, int8_t* h_output, int n) {
    int8_t* d_input;
    int8_t* d_output;

    // Allocate device memory
    hipMalloc(&d_input, n * sizeof(int8_t));
    hipMalloc(&d_output, (n / 2 + n % 2) * sizeof(int8_t));

    // Copy input data to device
    hipMemcpy(d_input, h_input, n * sizeof(int8_t), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (n + 2 * threadsPerBlock - 1) / (2 * threadsPerBlock);

    // Launch kernel
    quantize_array_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, n);

    // Copy result back to host
    hipMemcpy(h_output, d_output, (n / 2 + n % 2) * sizeof(int8_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}
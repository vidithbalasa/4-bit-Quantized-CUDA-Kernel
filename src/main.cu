#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int quantize(int8_t a, int8_t b = 0) {
    /*
    quantizes a pair of values to fit two 4-bit signed integers into a single byte
    */
    
    // input ints must fit in 4 bits
    if (a > 7 || a < -7 || b > 7 || b < -7) {
        printf("ERROR: quantize out of range\n");
        exit(1);
    }

    // Signed Ints: 2^3 & 2^8 ( [0]000[0]000 )
    int a_val = (a >> sizeof(int)-1) & 128;
    int8_t b_val = (b >> sizeof(int)-1) & 8;

    // Only care about values
    a = abs(a);
    b = abs(b);

    // Transalte a[4-bit] to left half of 8-bit int
    a_val += (a & 1) * 16;
    a_val += (a & 2) * 16;
    a_val += (a & 4) * 16;

    // Transalte b[4-bit] to right half of 8-bit int (last 3 bits)
    b_val += b & 7;

    // Combine a and b into a single byte
    return int8_t(a_val + b_val);
}

int[] split_ints(int x) {
    // int8_t a = x >> 4;
    // int8_t b = x & 15;

    // Get signed bit (bit 4)
    // int8_t a_sign = a >> 3;
    // int8_t a_val = (1 ** a_sign) * (-1 ** (1 - a_sign));
    // int8_t b_sign = b >> 3;
    // int8_t b_val = 1*b_sign + -1*1 - b_sign;

    // Get value (bits 0-3)
    // a_val += (a & 7);
    // b_val += (b & 7);

    // return [a_val, b_val];
    // return [a, b];
    return [0,0];
}

int8_t* split_ints(int8_t c, int8_t* split) {
    /*
    splits a single byte into two "4-bit" signed integers
    */

    // Split the byte into two 4-bit signed integers
    int8_t a = c >> 4;
    int8_t b = c & 15;

    // Get signed bit (bit 4)
    int8_t a_sign = a >> 3;
    int8_t a_val = (1 ** a_sign) * (-1 ** (1 - a_sign));
    int8_t b_sign = b >> 3;
    int8_t b_val = 1*b_sign + -1*1 - b_sign;

    // Get value (bits 0-3)
    a_val += (a & 7);
    b_val += (b & 7);

    // Add a and b to the list
    split[0] = a_val;
    split[1] = b_val;

    return split;
}


int main() {
    int8_t a = 6;
    int8_t b = -7;
    int8_t c = quantize(a, b);
    // printf("c =  %d\n", c);
    
    int8_t* split = new int8_t[2];
    split = split_ints(c, split);
    printf("a = %d\n", split[0]);
    printf("b = %d\n", split[1]);

    delete[] split;
    return 0;
}

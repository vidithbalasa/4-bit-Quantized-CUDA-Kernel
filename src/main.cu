#include <stdio.h>
#include <hip/hip_runtime.h>


#define MAX_4BIT 7
#define MIN_4BIT -7

__host__ __device__ int8_t quantize(int8_t a, int8_t b) {
    /* quantizes a pair of values to fit two 4-bit signed integers into a single byte */
    
    if (a > MAX_4BIT || a < MIN_4BIT || b > MAX_4BIT || b < MIN_4BIT) {
        printf("ERROR: Input integers must fit in 4 bits\n");
        return 0x80; // Set highest bit to indicate error
    }

    // Mask for value of signed ints: 2^3 & 2^8 ( [0]000[0]000 )
    int8_t a_val = a & 0x08;
    int8_t b_val = (b >> 4) & 0x08;

    // Masking for absolute value
    int8_t a_mask = a >> 7;
    a = (a ^ a_mask) - a_mask;
    int8_t b_mask = b >> 7;
    b = (b ^ b_mask) - b_mask;

    // transalte 4bit ints to left & right half of 8-bit int
    a_val += (a & 7) * 16;
    b_val += b & 7;

    return int8_t(a_val + b_val);
}

__global__ void quantize_array_kernel(const int8_t* input, int8_t* output, int n) {
    // Each thread will process two elements, so we calculate the index accordingly
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (2 * idx + 1 < n) { // Make sure we have both elements
        int8_t a = input[2 * idx];
        int8_t b = input[2 * idx + 1];
        output[idx] = quantize(a, b);
    }
}

void quantize_array(const int8_t* h_input, int8_t* h_output, int n) {
    int8_t* d_input;
    int8_t* d_output;

    // Allocate device memory
    hipMalloc(&d_input, n * sizeof(int8_t));
    hipMalloc(&d_output, (n / 2 + n % 2) * sizeof(int8_t));

    // Copy input data to device
    hipMemcpy(d_input, h_input, n * sizeof(int8_t), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    const int threadsPerBlock = 256;
    const int blocksPerGrid = (n + 2 * threadsPerBlock - 1) / (2 * threadsPerBlock);

    // Launch kernel
    quantize_array_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, n);

    // Copy result back to host
    hipMemcpy(h_output, d_output, (n / 2 + n % 2) * sizeof(int8_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}

void quantize_array_cpu(int8_t* arr, int8_t* quantized, int8_t size, int8_t* error) {
    int8_t index = 0;
    for (int i = 0; i < size-1; i += 2) {
        quantized[index] = quantize(arr[i], arr[i+1]);
        if (*error == 1) { return; }
        index += 1;
    }
    int8_t out_len = int(size / 2) + (size % 2);
    if (size % 2 == 1) {
        quantized[out_len-1] = quantize(arr[size-1], -0);
    }
}

void split_ints(int8_t c, int8_t* split) {
    /* splits a single byte into two "4-bit" signed integers */

    // split byte
    int8_t a = c >> 4;
    int8_t b = c & 15;

    // Get signed bit (4th bit from right)
    int8_t a_sign = a >> 3;
    int8_t a_multiplier = (1 * (1 - a_sign)) + (-1 * a_sign);

    int8_t b_sign = b >> 3;
    int8_t b_multiplier = (1 * (1 - b_sign)) + (-1 * b_sign);

    // Save 4bit int with multiplier
    split[0] = (a & 7) * a_multiplier;
    split[1] = (b & 7) * b_multiplier;
}

void unquantize_array(int8_t* arr, int8_t* unquantized, int8_t size) {
    int8_t index = 0;
    for (int i = 0; i < size; i++) {
        int8_t split[2];
        split_ints(arr[i], split);
        unquantized[index] = split[0];
        unquantized[index+1] = split[1];
        index += 2;
    }
}

int main() {
    int8_t error = 0;
    // Test Quantize
    int a_len = 1000;
    int8_t out_len = int(a_len / 2) + (a_len % 2);

    // create array of random ints between -7 and 7
    int8_t* arr = (int8_t*)malloc(a_len * sizeof(int8_t));
    for (int i = 0; i < a_len; i++) {
        arr[i] = rand() % 15 - 7;
    }

    //  Allocate 3 bytes of memory on host for int8 arrray
    int8_t out_arr[out_len];
    // quantize_array(arr, out_arr, a_len);

    // int8_t unquantized[a_len];
    // unquantize_array(out_arr, unquantized, a_len);

    int8_t out_arr_cpu[out_len];
    quantize_array_cpu(arr, out_arr_cpu, a_len, &error);

    // make sure there's no error
    if (error) {
        printf("Error found in main call");
    }

    // Make sure original array matches output array
    for (int i = 0; i < out_len; i++) {
        if (out_arr[i] != out_arr_cpu[i]) {
            printf("DIFFERS AT: %d != %d\n", out_arr[i], out_arr_cpu[i]);
        }
    }

    // printf("Quantized Array: ");
    // for (int i = 0; i < out_len; i++) {
    //     printf("%d ", out_arr[i]);
    // }
    // printf("\n");

    // printf("Unquantized Array: ");
    // for (int i = 0; i < a_len; i++) {
    //     printf("%d ", unquantized[i]);
    // }
    // printf("\n");

    free(arr);
    return 0;
}

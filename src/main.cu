#include <stdio.h>
#include <stdlib.h>
#include <cstdlib>
#include <hip/hip_runtime.h>


int quantize(int8_t a, int8_t b = 0) {
    /*
    quantizes a pair of values to fit two 4-bit signed integers into a single byte
    */
    
    // input ints must fit in 4 bits
    if (a > 7 || a < -7 || b > 7 || b < -7) {
        printf("ERROR: quantize out of range\n");
        exit(1);
    }

    // Signed Ints: 2^3 & 2^8 ( [0]000[0]000 )
    int a_val = (a >> sizeof(int)-1) & 128;
    int8_t b_val = (b >> sizeof(int)-1) & 8;

    // Only care about values
    a = abs(a);
    b = abs(b);

    // Transalte a[4-bit] to left half of 8-bit int
    a_val += (a & 1) * 16;
    a_val += (a & 2) * 16;
    a_val += (a & 4) * 16;

    // Transalte b[4-bit] to right half of 8-bit int (last 3 bits)
    b_val += b & 7;

    // Combine a and b into a single byte
    return int8_t(a_val + b_val);
}

int main() {
    int8_t a = 6;
    int8_t b = -7;
    int8_t c = quantize(a, b);
    printf("c = %d\n", int(sizeof(c)));

    return 0;
}
